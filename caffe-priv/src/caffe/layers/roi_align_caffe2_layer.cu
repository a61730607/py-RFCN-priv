#include "hip/hip_runtime.h"
#include <cfloat>
#include <algorithm>
#include <vector>

#include "caffe/layers/roi_align_caffe2_layer.hpp"
using std::max;
using std::min;

namespace caffe {

template <typename Dtype>
__device__ Dtype bilinear_interpolate(
    const Dtype* bottom_data,
    const int height,
    const int width,
    Dtype y,
    Dtype x,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (Dtype)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (Dtype)x_low;
  } else {
    x_high = x_low + 1;
  }

  Dtype ly = y - y_low;
  Dtype lx = x - x_low;
  Dtype hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  Dtype v1 = bottom_data[y_low * width + x_low];
  Dtype v2 = bottom_data[y_low * width + x_high];
  Dtype v3 = bottom_data[y_high * width + x_low];
  Dtype v4 = bottom_data[y_high * width + x_high];
  Dtype w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  Dtype val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename Dtype>
__global__ void ROIAlignForward(const int nthreads, const Dtype* bottom_data,
    const Dtype spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int sampling_ratio,
    const Dtype* bottom_rois, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const Dtype* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    Dtype roi_start_w = offset_bottom_rois[1] * spatial_scale;
    Dtype roi_start_h = offset_bottom_rois[2] * spatial_scale;
    Dtype roi_end_w = offset_bottom_rois[3] * spatial_scale;
    Dtype roi_end_h = offset_bottom_rois[4] * spatial_scale;

    Dtype roi_width = max(roi_end_w - roi_start_w, (Dtype)1.);
    Dtype roi_height = max(roi_end_h - roi_start_h, (Dtype)1.);

    Dtype bin_size_h = static_cast<Dtype>(roi_height) / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w = static_cast<Dtype>(roi_width) / static_cast<Dtype>(pooled_width);

    const Dtype* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);
    
    const Dtype count = roi_bin_grid_h * roi_bin_grid_w;

    Dtype output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const Dtype y = roi_start_h + ph * bin_size_h +
          static_cast<Dtype>(iy + .5f) * bin_size_h /
              static_cast<Dtype>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const Dtype x = roi_start_w + pw * bin_size_w +
            static_cast<Dtype>(ix + .5f) * bin_size_w /
                static_cast<Dtype>(roi_bin_grid_w);

        Dtype val = bilinear_interpolate(
            offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
    
  }
}

template <typename Dtype>
void ROIAlignCaffe2Layer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  
  if (bottom.size() > 2) {
      const Dtype* scale_pred = bottom[2]->gpu_data();
      caffe_gpu_asum<Dtype>(1, scale_pred, &spatial_scale_);
  }
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ROIAlignForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, spatial_scale_, channels_, height_, width_,
      pooled_height_, pooled_width_, sampling_ratio_, bottom_rois, top_data);
  CUDA_POST_KERNEL_CHECK;
}
template <typename Dtype>
inline __device__ Dtype gpu_atomic_add(const Dtype val, Dtype* address);

template <>
inline __device__ float gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

template <typename Dtype>
__device__ void bilinear_interpolate_gradient(
    const int height,
    const int width,
    Dtype y,
    Dtype x,
    Dtype& w1,
    Dtype& w2,
    Dtype& w3,
    Dtype& w4,
    int& x_low,
    int& x_high,
    int& y_low,
    int& y_high,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (Dtype)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (Dtype)x_low;
  } else {
    x_high = x_low + 1;
  }

  Dtype ly = y - y_low;
  Dtype lx = x - x_low;
  Dtype hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = bottom_data[y_low * width + x_low];
  // T v2 = bottom_data[y_low * width + x_high];
  // T v3 = bottom_data[y_high * width + x_low];
  // T v4 = bottom_data[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

template <typename Dtype>
__global__ void ROIAlignBackward(
    const int nthreads, const Dtype* top_diff, const int num_rois, const Dtype spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, 
    const int sampling_ratio,
    Dtype* bottom_diff,
    const Dtype* bottom_rois) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, h, w) coords in bottom data
    int pw = index % width;
    int ph = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    const Dtype* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    Dtype roi_start_w = offset_bottom_rois[1] * spatial_scale;
    Dtype roi_start_h = offset_bottom_rois[2] * spatial_scale;
    Dtype roi_end_w = offset_bottom_rois[3] * spatial_scale;
    Dtype roi_end_h = offset_bottom_rois[4] * spatial_scale;
    // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    Dtype roi_width = max(roi_end_w - roi_start_w, (Dtype)1.);
    Dtype roi_height = max(roi_end_h - roi_start_h, (Dtype)1.);
    Dtype bin_size_h = static_cast<Dtype>(roi_height) / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w = static_cast<Dtype>(roi_width) / static_cast<Dtype>(pooled_width);

    Dtype* offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const Dtype* offset_top_diff = top_diff + top_offset;
    const Dtype top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const Dtype count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const Dtype y = roi_start_h + ph * bin_size_h +
          static_cast<Dtype>(iy + .5f) * bin_size_h /
              static_cast<Dtype>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const Dtype x = roi_start_w + pw * bin_size_w +
            static_cast<Dtype>(ix + .5f) * bin_size_w /
                static_cast<Dtype>(roi_bin_grid_w);

        Dtype w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(
            height,
            width,
            y,
            x,
            w1,
            w2,
            w3,
            w4,
            x_low,
            x_high,
            y_low,
            y_high,
            index);

        Dtype g1 = top_diff_this_bin * w1 / count;
        Dtype g2 = top_diff_this_bin * w2 / count;
        Dtype g3 = top_diff_this_bin * w3 / count;
        Dtype g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          gpu_atomic_add(
              static_cast<Dtype>(g1), offset_bottom_diff + y_low * width + x_low);
          gpu_atomic_add(
              static_cast<Dtype>(g2), offset_bottom_diff + y_low * width + x_high);
          gpu_atomic_add(
              static_cast<Dtype>(g3), offset_bottom_diff + y_high * width + x_low);
          gpu_atomic_add(
              static_cast<Dtype>(g4), offset_bottom_diff + y_high * width + x_high);
        } // if
      } // ix
    } // iy
  } // CUDA_KERNEL_LOOP
} // RoIAlignBackward


template <typename Dtype>
void ROIAlignCaffe2Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* bottom_data = bottom[0]->gpu_data();
  //auto& X = Input(0);
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  //auto& R = Input(1);
  const Dtype* top_diff = top[0]->gpu_diff();
  //auto& dY = Input(2);
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  //auto* dX = Output(0);
  
  caffe_gpu_set(bottom[0]->count(), Dtype(0.), bottom_diff);
  const int count = top[0]->count();

  if (bottom.size() > 2) {
      const Dtype* scale_pred = bottom[2]->gpu_data();
      caffe_gpu_asum<Dtype>(1, scale_pred, &spatial_scale_);
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  ROIAlignBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff,  bottom[1]->num(), spatial_scale_, channels_,
      height_, width_, pooled_height_, pooled_width_, sampling_ratio_, bottom_diff, bottom_rois);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(ROIAlignCaffe2Layer);

}
